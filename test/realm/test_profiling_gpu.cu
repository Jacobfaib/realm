#include <hip/hip_runtime.h>

#include <cstdint>

__global__ void spin_kernel(uint64_t t_ns)
{
  uint64_t start = 0, current = 0;
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(start));
  do {
    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(current));
  } while((current - start) < t_ns);
}

void launch_spin_kernel(uint64_t t_ns, hipStream_t stream)
{
  void *args[] = {&t_ns};
  hipError_t err = hipLaunchKernel(reinterpret_cast<const void*>(reinterpret_cast<void *>(spin_kernel)), dim3(1),
                                     dim3(1), args, 0, static_cast<hipStream_t>(stream));
}